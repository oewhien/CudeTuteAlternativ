#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;


__global__ void AddIntsCUDA(int* a, int* b) {
	for (int i = 0; i < 12000000; i++)
	{
		a[0] += b[0];
	}
	
}


int main() {


	int a = 0;
	int b = 1;

	int *d_a, *d_b;

	hipMalloc(&d_a, sizeof(int));
	hipMalloc(&d_b, sizeof(int));

	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

	AddIntsCUDA<<<1, 1>>>(d_a, d_b);

	hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);

	cout << "The answer is " << a << endl;
	cin >> "";

	hipFree(d_a);
	hipFree(d_b);



	return 0;
}
